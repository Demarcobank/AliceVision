#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// Copyright (c) 2022 AliceVision contributors.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

#include "deviceFuse.hpp"
#include "deviceFuseKernels.cuh"

#include <aliceVision/depthMap/cuda/hostUtils.hpp>

namespace aliceVision {
namespace depthMap {

__host__ void cuda_fuseDepthSimMapsGaussianKernelVoting(CudaDeviceMemoryPitched<float2, 2>& out_depthSimMapRefinedFused_dmp,
                                                        const CudaDeviceMemoryPitched<float2, 2>& in_depthSimMapPartSgmUpscale_dmp,
                                                        const std::vector<CudaDeviceMemoryPitched<float2, 2>>& in_depthSimMapPartPerRcTc_dmp,
                                                        const RefineParams& refineParams,
                                                        const ROI& roi, 
                                                        hipStream_t stream)
{
    const float samplesPerPixSize = float(refineParams.nSamplesHalf / ((refineParams.nDepthsToRefine - 1) / 2));
    const float twoTimesSigmaPowerTwo = 2.0f * refineParams.sigma * refineParams.sigma;

    // setup block and grid
    const int blockSize = 16;
    const dim3 block(blockSize, blockSize, 1);
    const dim3 grid(divUp(roi.width(), blockSize), divUp(roi.height(), blockSize), 1);

    const CudaSize<2> roiSize(roi.width(), roi.height());

    assert(roiSize == in_depthSimMapPartPerCam_dmp.front().getSize());

    CudaDeviceMemoryPitched<float2, 2> bestGsvSampleMapPart_dmp(roiSize);
    CudaDeviceMemoryPitched<float, 2> gsvSampleMapPart_dmp(roiSize);

    // sliding gaussian window
    for(int sample = -refineParams.nSamplesHalf; sample <= refineParams.nSamplesHalf; ++sample) // default sample range from -150 to 150
    {
        // compute the gaussian window sample 
        for(int tci = 0; tci < in_depthSimMapPartPerRcTc_dmp.size(); ++tci) // number of T cameras
        {
            // sum gaussian window sample score of each RcTc depth/sim map
            fuse_computeGaussianKernelVotingSampleMap_kernel<<<grid, block, 0, stream>>>(
                gsvSampleMapPart_dmp.getBuffer(), 
                gsvSampleMapPart_dmp.getPitch(), 
                in_depthSimMapPartPerRcTc_dmp[tci].getBuffer(), // tc depth/sim map 
                in_depthSimMapPartPerRcTc_dmp[tci].getPitch(),
                in_depthSimMapPartSgmUpscale_dmp.getBuffer(), // sgm depth/pixSize map for middle depth
                in_depthSimMapPartSgmUpscale_dmp.getPitch(),
                tci, // first tc cam id, (re)-initialization
                float(sample),
                samplesPerPixSize, 
                twoTimesSigmaPowerTwo,
                roi);
        }

        // save the sample if it's the best
        fuse_updateBestGaussianKernelVotingSampleMap_kernel<<<grid, block, 0, stream>>>(
            bestGsvSampleMapPart_dmp.getBuffer(), 
            bestGsvSampleMapPart_dmp.getPitch(), 
            gsvSampleMapPart_dmp.getBuffer(),
            gsvSampleMapPart_dmp.getPitch(), 
            sample + refineParams.nSamplesHalf, // first sample, first initialization 
            float(sample),                 
            roi);
    }

    // write the output depth/sim for the best sample
    fuse_computeFusedDepthSimMapFromBestGaussianKernelVotingSampleMap_kernel<<<grid, block, 0, stream>>>(
        out_depthSimMapRefinedFused_dmp.getBuffer(), 
        out_depthSimMapRefinedFused_dmp.getPitch(), 
        bestGsvSampleMapPart_dmp.getBuffer(),
        bestGsvSampleMapPart_dmp.getPitch(), 
        in_depthSimMapPartSgmUpscale_dmp.getBuffer(), // sgm depth/pixSize map for middle depth
        in_depthSimMapPartSgmUpscale_dmp.getPitch(), 
        samplesPerPixSize,
        roi);

    CHECK_CUDA_ERROR();
}

__host__ void cuda_optimizeDepthSimMapGradientDescent(CudaDeviceMemoryPitched<float2, 2>& out_depthSimMapOptimized_dmp,
                                                      const CudaDeviceMemoryPitched<float2, 2>& in_depthSimMapSgmUpscale_dmp,
                                                      const CudaDeviceMemoryPitched<float2, 2>& in_depthSimMapRefinedFused_dmp,
                                                      const DeviceCamera& rcDeviceCamera, 
                                                      const RefineParams& refineParams,
                                                      const ROI& roi,
                                                      hipStream_t stream)
{
    const float samplesPerPixSize = float(refineParams.nSamplesHalf / ((refineParams.nDepthsToRefine - 1) / 2));

    // initialize depth/sim map optimized with SGM depth/sim map
    copy(out_depthSimMapOptimized_dmp, in_depthSimMapSgmUpscale_dmp);
    
    const CudaSize<2> roiSize(roi.width(), roi.height());
    CudaDeviceMemoryPitched<float, 2> optDepthMapPart_dmp(roiSize);
    CudaDeviceMemoryPitched<float, 2> imgVariancePart_dmp(roiSize);

    {
        // setup block and grid
        const dim3 lblock(32, 2, 1);
        const dim3 lgrid(divUp(roi.width(), lblock.x), divUp(roi.height(), lblock.y), 1);

        compute_varLofLABtoW_kernel<<<lgrid, lblock, 0, stream>>>(
            rcDeviceCamera.getTextureObject(), 
            imgVariancePart_dmp.getBuffer(), 
            imgVariancePart_dmp.getPitch(),
            roi);
    }

    CudaTexture<float> imgVarianceTex(imgVariancePart_dmp);

    // setup block and grid
    const int blockSize = 16;
    const dim3 block(blockSize, blockSize, 1);
    const dim3 grid(divUp(roi.width(), blockSize), divUp(roi.height(), blockSize), 1);

    for(int iter = 0; iter < refineParams.nIters; ++iter) // default nb iterations is 100
    {
        // copy depths values from optDepthSimMap to optDepthMap
        fuse_getOptDeptMapFromOptDepthSimMap_kernel<<<grid, block, 0, stream>>>(
            optDepthMapPart_dmp.getBuffer(), 
            optDepthMapPart_dmp.getPitch(), 
            out_depthSimMapOptimized_dmp.getBuffer(), // initialized with SGM depth/sim map
            out_depthSimMapOptimized_dmp.getPitch(),
            roi);

        CudaTexture<float> depthTex(optDepthMapPart_dmp);

        // adjust depth/sim by using previously computed depths
        fuse_optimizeDepthSimMap_kernel<<<grid, block, 0, stream>>>(
            rcDeviceCamera.getTextureObject(), 
            rcDeviceCamera.getDeviceCamId(), 
            imgVarianceTex.textureObj,
            depthTex.textureObj, 
            out_depthSimMapOptimized_dmp.getBuffer(), 
            out_depthSimMapOptimized_dmp.getPitch(),
            in_depthSimMapSgmUpscale_dmp.getBuffer(), 
            in_depthSimMapSgmUpscale_dmp.getPitch(),
            in_depthSimMapRefinedFused_dmp.getBuffer(), 
            in_depthSimMapRefinedFused_dmp.getPitch(),
            iter, 
            samplesPerPixSize, 
            roi);
    }

    CHECK_CUDA_ERROR();
}

} // namespace depthMap
} // namespace aliceVision
